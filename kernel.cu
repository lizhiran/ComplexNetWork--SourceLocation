#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
//#include "common_functions.h"
//#include <hip/hip_runtime.h>
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <dos.h>
#include "GuassRandom.h"
#include "graph.h"
#include "SqQueue.h"
#define BLOCK_DIM 16  //COUNT-1��ֵ���ΪBLOCK_DIM ��������  
//4ʱ�����13���۲��       4*4���߳̿�
//8ʱ�����25���۲�� �����24*24�ľ���  ��Ӳ�����ƣ�512���̣߳����3*3���߳̿�         
//9  2*2 19
//10 2*2 21
//16 2*2 33
//20 1*1 21
#define ARRIVAL_TIME 199
#include "hip/device_functions.h"


int *activeObservers;
int COUNT = -1;

hipError_t MatrixWithCuda(float *csFinal,float *csEnd, int height,int width);
//
//__global__ void cuda_Matrix(float* odata, float* idata, int width, int height)
//{
//	__shared__ float block[BLOCK_DIM][BLOCK_DIM+1];
//	// read matrix tile into shared memory
//	unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
//	unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
//	if ((xIndex < width ) && (yIndex < height))
//	{
//		unsigned int index_in = yIndex * width + xIndex;
//		block[threadIdx.y][threadIdx.x] = idata[index_in];
//	}
//
//	__syncthreads();
//	// write the transposed matrix tile to global memory
//	xIndex = blockIdx.y * blockDim.y + threadIdx.x;
//	yIndex = blockIdx.x * blockDim.x + threadIdx.y;
//	if((xIndex < height) && (yIndex < width))
//	{
//		unsigned int index_out = yIndex * height + xIndex;
//		odata[index_out] = block[threadIdx.x][threadIdx.y];
//	}
//}
// ��������֮��������
__global__ void MatrixInverse_Elimination(float *A,float *B,int n,int i)
{
	//����洢�����
	__shared__ float As[BLOCK_DIM][BLOCK_DIM];
	__shared__ float Bs[BLOCK_DIM][BLOCK_DIM];
	int tix = blockDim.x * blockIdx.x + threadIdx.x;
	int tiy = blockDim.y * blockIdx.y + threadIdx.y;

	if(tiy<n && tix<n)
	{
		//����block������洢��
		As[threadIdx.y][threadIdx.x] = A[tiy*n+tix];
		Bs[threadIdx.y][threadIdx.x] = B[tiy*n+tix];
		__syncthreads();

			float a = A[tiy*n+i]*A[i*n+tix];
			float b = A[tiy*n+i]*B[i*n+tix];
			
			//�ù�һ���ĵ�i�����i������
			As[threadIdx.y][threadIdx.x] = As[threadIdx.y][threadIdx.x] - a;
			Bs[threadIdx.y][threadIdx.x] = Bs[threadIdx.y][threadIdx.x] - b;

		__syncthreads();
		if(tiy != i)        //��������������Դ�
		{
			A[tiy*n+tix] = As[threadIdx.y][threadIdx.x];
			B[tiy*n+tix] = Bs[threadIdx.y][threadIdx.x];
		}
		//__syncthreads();
	}
}
__global__ void MatrixInverse_Normalized(float *A,float *B,int n,int i)
{
	int tix = blockDim.x * blockIdx.x + threadIdx.x;
	int tiy = blockDim.y * blockIdx.y + threadIdx.y;
	float temp;//��һ��ֵ
	temp = A[i*n+i];
	//��A,B�����i������һ������
	if(tiy<n && tix<n)
	{
		if (tix == i)
		{
		
			A[tix*n + tiy] /=temp;
			B[tix*n + tiy] /=temp;
		}
		//__syncthreads();
	}
}


void randomSetWeight(ALGraph *graph)//���Խ��˲���ŵ�����ͼ�Ĺ�����
{
	float a[250] = {11.0,10.0,17.0,14.0,7.0,9.0,8.0,12.0,13.0,8.0,20.0,7.0,12.0,9.0,14.0,8.0,12.0,12.0,13.0,16.0,7.0,13.0,10.0,9.0,12.0,12.0,15.0,10.0,13.0,12.0,14.0,17.0,14.0,8.0,12.0,12.0,12.0,7.0,10.0,12.0,12.0,12.0,9.0,10.0,12.0,10.0,5.0,14.0,10.0,10.0,6.0,10.0,11.0,10.0,16.0,11.0,14.0,15.0,12.0,12.0,13.0,12.0,10.0,12.0,13.0,11.0,7.0,8.0,7.0,10.0,9.0,13.0,10.0,15.0,11.0,15.0,12.0,12.0,11.0,10.0,14.0,14.0,10.0,14.0,11.0,6.0,14.0,10.0,9.0,12.0,13.0,9.0,12.0,14.0,12.0,11.0,13.0,12.0,13.0,9.0,20.0,7.0,15.0,11.0,11.0,14.0,13.0,14.0,12.0,10.0,12.0,19.0,11.0,14.0,8.0,9.0,11.0,8.0,10.0,9.0,7.0,10.0,14.0,7.0,13.0,13.0,4.0,8.0,5.0,10.0,5.0,8.0,8.0,7.0,13.0,15.0,15.0,14.0,11.0,12.0,12.0,7.0,7.0,8.0,8.0,10.0,6.0,8.0,8.0,15.0,15.0,12.0,12.0,11.0,8.0,10.0,7.0,15.0,13.0,7.0,16.0,7.0,14.0,12.0,18.0,11.0,9.0,11.0,8.0,14.0,15.0,13.0,10.0,12.0,11.0,9.0,13.0,10.0,9.0,7.0,12.0,8.0,13.0,11.0,10.0,13.0,11.0,13.0,11.0,12.0,10.0,11.0,16.0,13.0,7.0,11.0,10.0,14.0,10.0,15.0,5.0,15.0,11.0,6.0,14.0,9.0,11.0,12.0,11.0,9.0,16.0,17.0,11.0,10.0,13.0,6.0,8.0,15.0,12.0,13.0,16.0,14.0,15.0,6.0,10.0,11.0,7.0,12.0,10.0,12.0,11.0,15.0,9.0,17.0,6.0,7.0,10.0,16.0,10.0,8.0,9.0,9.0,9.0,12.0,13.0,9.0,17.0,12.0,7.0,13.0};

	for(int i =0; i< graph->arcnum; i++)
	{
		//graph->arctices[i].Weight = (int)(3*GetOneGaussian(0,1.0)+12);
		//graph->arctices[i].tmpWeight = graph->arctices[i].Weight;
		 		graph->arctices[i].Weight = a[i];
		 		graph->arctices[i].tmpWeight = a[i];
		//	printf("%d+%d\n" ,i,graph->arctices[i].Weight);
	}
}
void setAttribute(ALGraph *graph)
{
	for (int i =0 ;i<graph->vexnum;i++)
	{
		graph->vertices[i].time = -1;
		graph->vertices[i].isActive = false;
		graph->vertices[i].direction = -1;
	}
}

//int diffArray[DIFF_EDGE_NUM];
int *diffArray;
static int diffLength = 0;
void help_add(int arcNode)
{
	diffArray[diffLength] = arcNode;
	diffLength ++;
}
void help_remove(int temp)
{
	for(int j=0;j<diffLength;j++)
		if(diffArray[j]==temp)
		{
			diffArray[j]=-1;
			break;
		}     
}
bool help_contain(int temp)
{

	for(int j=0;j<diffLength;j++)
	{
		if(diffArray[j]==temp)
			return true;
	}
	return false;
}
/************************************************************************/
/* ��Ϣ����                                                                     */
/************************************************************************/
void diffusion(ALGraph *graph, int source)//sourceΪrealNode�Ľڵ�ID
{
	int time = ARRIVAL_TIME;//199
	int zerotime = 1;
	int sourceNodeIndex = GetNodeIndex(*graph,source);
	int sourceDegree = graph->vertices[sourceNodeIndex].degree;
	//	int *edges;
	//	edges = (int *)malloc(sourceDegree * sizeof(int));
	//int edges[sourceDegree]; 
	int *edges = getNodeEdges(graph,source);
	int i;
	//     for (i =0 ;i<sourceDegree;i++)
	//     {
	// 		printf("%d ",edges[i]);//Դ�ڵ���ٱ�:�ߵ�ID
	//     }
	diffLength = 0;
	if (diffArray != NULL)
	{
		free(diffArray);
	}
	diffArray = (int *)malloc((graph->arcnum)*sizeof(int));
	for (i=0;i<sourceDegree;i++)
	{
		help_add(edges[i]);
	}

	graph->vertices[sourceNodeIndex].time=1;
	graph->vertices[sourceNodeIndex].direction=-1;
	graph->vertices[sourceNodeIndex].isActive=true;

	while(zerotime<time)
	{
		for (i=0;i<diffLength;i++)
		{
			if (diffArray[i]!=-1)
			{
				//diffArray[i]�е�ֵΪ�ߵ�ID��Ҳ�Ǳ���graph�е���������
				//edgeID = diffArray[i] = arc.arcid
				Arc  *arc= &(graph->arctices[diffArray[i]]);
				int weight = arc->tmpWeight;
				if (weight-1==0)
				{
					int s = arc->arcsourceId;
					int t = arc->arctargerId;
					if (!(graph->vertices[GetNodeIndex(*graph,t)].isActive))
					{
						graph->vertices[GetNodeIndex(*graph,t)].isActive = true;
						graph->vertices[GetNodeIndex(*graph,t)].direction = s;
						graph->vertices[GetNodeIndex(*graph,t)].time = zerotime+1;
						int *tids = getNodeEdges(graph,t);
						int tdegree = graph->vertices[GetNodeIndex(*graph,t)].degree;
						for (int j=0;j<tdegree;j++)
						{
							Arc *arct =&(graph->arctices[tids[j]]);
							int tnode = arct->arcsourceId;
							if (tnode == t)
							{
								tnode = arct->arctargerId;
							} 
							if (!help_contain(arct->arcId) && !(graph->vertices[GetNodeIndex(*graph,tnode)].isActive))
							{
								help_add(arct->arcId);
								arct->tmpWeight = arct->tmpWeight + 1;
							}
						}
						help_remove(arc->arcId);
					} 
					else if(!(graph->vertices[GetNodeIndex(*graph,s)].isActive))
					{
						graph->vertices[GetNodeIndex(*graph,s)].isActive = true;
						graph->vertices[GetNodeIndex(*graph,s)].direction = t;
						graph->vertices[GetNodeIndex(*graph,s)].time = zerotime+1;
						int *sids = getNodeEdges(graph,s);
						int sdegree = graph->vertices[GetNodeIndex(*graph,s)].degree;
						for (int j=0;j<sdegree;j++)
						{
							Arc *arcs = &(graph->arctices[sids[j]]);
							int snode = arcs->arcsourceId;
							if (snode == s)
							{
								snode = arcs->arctargerId;
							} 
							if (!help_contain(arcs->arcId) && !(graph->vertices[GetNodeIndex(*graph,snode)].isActive))
							{
								help_add(arcs->arcId);
								arcs->tmpWeight = arcs->tmpWeight + 1;
							}
						}
						help_remove(arc->arcId);
					}else
					{
						help_remove(arc->arcId);
					}
				}
				arc->tmpWeight = arc->tmpWeight -1;
			}
		}
		zerotime++;
	}
	free(edges);
}
bool isarray_contain(int *tempArray,int temp)//���������������ȥ��
{

	for(int j=0;j<COUNT;j++)
	{
		if(tempArray[j]==temp)
			return true;//����temp����true
	}
	return false;
}
/************************************************************************/
/* ���ѡ��۲�����                 */
/************************************************************************/
void selectObserversAsRandom(ALGraph graph)
{
	if (activeObservers != NULL)
	{
		free(activeObservers);
	}
	activeObservers = (int *)malloc(COUNT*sizeof(int));
	int *temp = (int *)malloc(COUNT*sizeof(int));
	int i = 0;
	while(i<COUNT)
	{
		int randIndex = rand()%graph.vexnum;
		if (!isarray_contain(temp,randIndex)&&graph.vertices[randIndex].isActive)
		{
			temp[i] = randIndex;
			//	printf("temp[%d]=%d \n",i,temp[i]);
			//	activeObservers[i] = i+1;
			activeObservers[i] = graph.vertices[randIndex].nodeid;
			i++;
			//printf("nodeId %d\n",activeObservers[i]);
		}
	}
	// 	for (i =0;i<COUNT;i++)
	// 	{
	// 		//printf("%d\n",activeObservers[i]);
	// 	}
	free(temp);
}
/************************************************************************/
/* ���ն����в���ѡ��۲��  */
/* ����������ѡ�񣬰���С��ѡ��*/ //����Ч������С��ѡ��ȽϺ�
/************************************************************************/
void selectObserversAsDegree(ALGraph graph)
{
	if (activeObservers != NULL)
	{
		free(activeObservers);
	}
	activeObservers = (int *)malloc(COUNT*sizeof(int));
	int *temp = (int *)malloc(COUNT*sizeof(int));
	int maxDegree = 0; 
	//int minDegree = 1000;
	int maxDegreeNodeID = 0;
	//for (int i=0; i<COUNT;i++)
	int i = 0;
//	int num =0;
	while(i<COUNT)
	{

		for (int j=0; j<graph.vexnum; j++)
		{
			//if (!isarray_contain(temp,graph.vertices[j].nodeid) && graph.vertices[j].degree<maxDegree)
			if (!isarray_contain(temp,graph.vertices[j].nodeid) && graph.vertices[j].degree>maxDegree)
			{

				maxDegreeNodeID = graph.vertices[j].nodeid;
				maxDegree = graph.vertices[j].degree;
				//minDegree = graph.vertices[j].degree;
			}
		}	
		if (graph.vertices[GetNodeIndex(graph,maxDegreeNodeID)].isActive)
		{
			temp[i] = maxDegreeNodeID;
			activeObservers[i] = maxDegreeNodeID;
			i++;
		}	
		maxDegree = 0;
		//	minDegree = 1000;
		maxDegreeNodeID = 0;
	}
	printf("�۲��Ϊ��\n");
	for (i =0;i<COUNT;i++)
	{
		printf("%d\n",activeObservers[i]);
	}
	free(temp);
}
void setObserver(ALGraph *graph)
{
	for (int i=0; i<graph->arcnum; i++)
	{
		graph->arctices[i].tmpWeight = graph->arctices[i].Weight;
	}

}
void propagation(ALGraph *graph,int sourceid)
{
	setAttribute(graph);
	diffusion(graph,sourceid);
	//	selectObserversAsRandom(*graph);
	selectObserversAsDegree(*graph);
	setObserver(graph);// ��ҪĿ���ǽ�tmpweight��ԭ������һ��ѭ��ʹ��
}
bool isset_contain(int *tempArray,int temp,int arraylenth)//
{

	for(int j=0;j<arraylenth;j++)
	{
		if(tempArray[j]==temp)
			return true;
	}
	return false;
}
void generatorBFSTress(ALGraph *graph,int rootId)
{
	SqQueue Q;
	InitQueue(Q);
	int *set;
	// 	if (set !=NULL )
	// 	{
	// 		free(set);
	// 	}
	set = (int *)malloc((graph->vexnum)*sizeof(int));
	for (int i=0 ;i<graph->vexnum;i++)
	{
		set[i] = -1;
	}
	int  len;
	graph->vertices[GetNodeIndex(*graph,rootId)].parent = -1;
	graph->vertices[GetNodeIndex(*graph,rootId)].plength = 0;
	set[0]  = rootId;
	int setI = 1;
	EnQueue(Q,rootId);
	while(!QueueEmpty(Q))
	{
		int qNodeId ;
		DeQueue(Q,qNodeId);
		len = graph->vertices[GetNodeIndex(*graph,qNodeId)].plength;

		int *neighbors = getNodeEdges(graph,qNodeId);//������ÿ�����ڵıߣ��༴����������ڵ����ٵĽڵ㡣���رߵ�ID
		for (int n =0; n<graph->vertices[GetNodeIndex(*graph,qNodeId)].degree; n++)
		{
			int nId = -1; // �õ�һ�����ڵĽڵ�ID
			if (graph->arctices[neighbors[n]].arcsourceId==qNodeId)
			{
				nId = graph->arctices[neighbors[n]].arctargerId;
			} 
			else if (graph->arctices[neighbors[n]].arctargerId==qNodeId)
			{
				nId = graph->arctices[neighbors[n]].arcsourceId;
			}
			if (!isset_contain(set,nId,graph->vexnum))
			{
				set[setI] = nId;
				setI++;
				EnQueue(Q,nId);
				graph->vertices[GetNodeIndex(*graph,nId)].parent = qNodeId;
				graph->vertices[GetNodeIndex(*graph,nId)].plength = len +1;

			}

		}
	}
	free(set);
	DestroyQueue(Q);
}
void cMatrix(float *a, int n, float *end)
{
	//	float Vp[][] = new float[n + 1][2 * n + 1];
	float **Vp;
	int xi,yj;
	Vp = (float **)malloc(sizeof(float *)*(n+1));
	for (int vpi=0;vpi<n+1;vpi++)
	{
		Vp[vpi] = (float *) malloc((sizeof(float)*(2*n+1)));
	}
	for (xi=0; xi<n+1; xi++)
	{
		for (yj=0; yj<2*n+1; yj++)
		{
			Vp[xi][yj] =0.0;//Ҫ�Գ�ʼ�����󸳳�ֵ������C�ĳ�ֵ��Ϊ0
		}
	}
	int i, j,k;
	i = j = n;
	// 	for ( k = 1; k < n + 1; k++) {
	// 		for (int t = 1; t < 2 * n + 1; t++) {
	// 			Vp[k][t] = 0.0;
	// 		}
	// 	}
	//input the data of the matrix
	for (k = 1; k <= n; k++) {
		for (int t = 1; t <= n; t++) {
			Vp[k][t] = a[(k - 1)*n+t - 1];
		}
	}


	for (k = 1; k <= i; k++) {
		for (int t = j + 1; t <= j * 2; t++) {
			if ((t - k) == j) {
				Vp[k][t] = 1.0;
			} else {
				Vp[k][t] = 0;
			}
		}
	}

	for (k = 1; k <= i; k++) {
		if (Vp[k][k] != 1) {
			float bs = Vp[k][k];
			Vp[k][k] = 1;
			for (int p = k + 1; p <= j * 2; p++) {
				Vp[k][p] /= bs;
			}
		}

		for (int q = 1; q <= i; q++) {
			if (q != k) {
				float bs = Vp[q][k];
				for (int p = 1; p <= j * 2; p++) {
					Vp[q][p] -= bs * Vp[k][p];
				}
			} else {
				continue;
			}
		}
	}
	//print out the result of the change
	//  System.out.println("---------------\nThe result is:");
	for (int x = 1; x <= i; x++) {
		for (int y = j + 1; y <= j * 2; y++) {
			//a[x-1][y-2]=Vp[x][y];


			//end[x - 1][y - j - 1] = Vp[x][y];
			end[(x-1)*n+y-j-1] = Vp[x][y];

			//System.out.print(Vp[x][y] + "  ");
		}
		//System.out.println();
	}
}
int main()
{
	printf("**************������۲��ĸ���*****************\n");
	printf("COUNT>");
	scanf("%d",&COUNT);
	if (COUNT <=0)
	{
		printf("COUNT<=0");
		exit(0);
	}
	ALGraph graph;
	CreateGraph(&graph);
	Display(graph);
	srand((unsigned)time(NULL));//���������Ӧ��
	randomSetWeight(&graph);

	int hit = 0;
	//��һ��for�����нڵ㣬������
	for (int i =0; i<graph.vexnum;i++)//�����ڵ����������ͼҪΪ�����ͨ��ͼ
	{

		int realSourceNodeId = graph.vertices[i].nodeid;
		propagation(&graph,realSourceNodeId);

		//���Ƴ���ǰ�ڵ�ΪԴ��ʱ�����ô������̣�ѡ��һ�ֲ�����ԣ�������Ƶ����ֵ�Ľڵ�ID������������ֵ��Դ���λ�ò�࣬���������
		float maxEstimator = -100000;
		int maxNodeId = -1;
		//	printf("\n********�ӳ�������ֵ***********\n");
		int activeObserversSize = COUNT;
		int *delay = (int*)malloc((activeObserversSize-1)*sizeof(int));//activeObseversize ΪCOUNT-1
		int refObseverIndex = GetNodeIndex(graph,activeObservers[0]);
		int reftime = graph.vertices[refObseverIndex].time;

		for (int delayi=1; delayi<COUNT; delayi++)
		{	
			int curObseverIndex = GetNodeIndex(graph,activeObservers[delayi]);
			delay[delayi-1] = graph.vertices[curObseverIndex].time - reftime;
			// printf("%d\n",delay[delayi-1]);	//�ӳ�����d��ֵ
		}

		for (int exti=0; exti<graph.vexnum;exti++)
		{
			//printf("BFS Start\n");
			generatorBFSTress(&graph,graph.vertices[exti].nodeid);//����BFS��
			//printf("BFS End\n");
			float *us = (float *)malloc((activeObserversSize-1)*sizeof(float));
			float *usT = (float *)malloc((activeObserversSize-1)*sizeof(float));
			float refus = graph.vertices[refObseverIndex].plength;
			//printf("us Start");
			//	printf("%f\n",us[0]);
			for (int oi = 1; oi<COUNT; oi++)
			{
				int curObseverIndex = GetNodeIndex(graph,activeObservers[oi]);//activeObserversΪ�ڵ�IDֵ
				//	printf("%f\n",graph.vertices[curObseverIndex].plength);
				us[oi-1] = (graph.vertices[curObseverIndex].plength - refus)*0.5;
				usT[oi - 1] = us[oi - 1] * 2 * 12;
				//	printf("%f\n",us[oi-1]);

			}
			for (int j=0 ; j<activeObserversSize-1;j++)
			{
				us[j] = delay[j] - us[j]*12;//d-0.5*us
			}
		//	printf("us end");
			int **cs;
			cs = (int **)malloc(sizeof(int *)*activeObserversSize);
			//cs[0] = (int *) malloc((sizeof(int)*activeObserversSize*graph.arcnum));
			for (int csi=0;csi<activeObserversSize;csi++)
			{
				cs[csi] =  (int *) malloc((sizeof(int)*graph.arcnum));
			}
			int x,y;
			for (x=0; x<activeObserversSize; x++)
			{
				for (y=0; y<graph.arcnum; y++)
				{
					cs[x][y] =0;//Ҫ�Գ�ʼ�����󸳳�ֵ������C�ĳ�ֵ��Ϊ0
				}
			}
			for (int k=0; k<activeObserversSize; k++)//����CS������ÿ���۲�㵽Դ���·��
			{
				int observerNodeID = activeObservers[k];
				while(graph.vertices[GetNodeIndex(graph,observerNodeID)].parent != -1)
				{
					int parentid = graph.vertices[GetNodeIndex(graph,observerNodeID)].parent;
					int edgeid = GetEdgeId(graph,parentid,observerNodeID);
					cs[k][edgeid] = 1;
					observerNodeID = parentid;
				}
			}
			float *csFinal;
			csFinal = (float *)malloc(sizeof(float)*(activeObserversSize-1)*(activeObserversSize-1));
			for (x=0; x<(activeObserversSize-1)*(activeObserversSize-1); x++)
			{
				/*for (y=0; y<activeObserversSize-1; y++)
				{*/
					csFinal[x] =0.0;//Ҫ�Գ�ʼ�����󸳳�ֵ������C�ĳ�ֵ��Ϊ0
				//}
			}
			float *csEnd;
			csEnd = (float *)malloc(sizeof(float)*(activeObserversSize-1)*(activeObserversSize-1));

		
			//�������ÿһ�м�ȥ��һ��
			int m,n;//ѭ������
			for(m=1; m<activeObserversSize;m++)
			{
				n=0;
				while(n < graph.arcnum)
				{
					cs[m][n] = cs[m][n] - cs[0][n];
					//	printf("%d ",cs[m][n]);
					n++;
				}
			}
			int sum = 0;
			for (m=1; m<activeObserversSize;m++)
			{
				for (n=1; n<activeObserversSize; n++)
				{
					int l = 0;
					while(l < graph.arcnum)
					{
						sum = sum + cs[n][l] * cs[m][l];
						l++;
					}
					//csFinal[m-1][n-1] = sum *9;
					csFinal[(m-1)*(activeObserversSize-1)+n-1] = (float)sum * 9;
					//	printf("%f ",csFinal[m-1][n-1]);
					sum = 0;
				}
			}
			//printf("csFinal End\n");
			//printf("Matrix Start\n");		
			// Add vectors in parallel.
			hipError_t cudaStatus = MatrixWithCuda(csFinal,csEnd,COUNT-1,COUNT-1);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "MatrixWithCuda failed!");
				return 1;
			}
		/*	printf("*******************************************\n");
			for (int x=0;x<COUNT-1;x++)
			{
				for (int y=0;y<COUNT-1;y++)
				{
					printf("%f ",csEnd[x*(COUNT-1)+y]);
				}
				printf("\n");
			}
			printf("*******************************************\n");
			for (int c=0;c<COUNT-1;c++)
			{
				for (int d=0;d<COUNT-1;d++)
				{
					printf("%f ",csFinal[c*(COUNT-1)+d]);
				}
				printf("\n");
			}
			printf("*******************************************\n");*/
			//cMatrix(csFinal,activeObserversSize-1,csEnd1);//�������
		/*	for (int x=0;x<COUNT-1;x++)
			{
				for (int y=0;y<COUNT-1;y++)
				{
					printf("%f ",csEnd1[x*(COUNT-1)+y]);
				}
				printf("\n");
			}
			printf("Matrix End\n");*/
			float *usTT = (float *)malloc((activeObserversSize-1)*sizeof(float));
			for (m=0; m<activeObserversSize-1; m++)
			{
				float s =0;
				for (n = 0; n < activeObserversSize-1; n++)
				{
					//s = s + usT[n] * csEnd[n][m];
					s = s + usT[n] * csEnd[n*(activeObserversSize-1)+m];
				}
				usTT[m] = s;
			}
			float es = 0.0;
			for (m=0; m<activeObserversSize-1; m++)
			{
				es = es+usTT[m]* us[m];
			}
			//	printf("\n****����Դ��Ϊ%dʱ���Ƶ����ֵΪ%f\n",graph.vertices[exti].nodeid,es);
			if (es > maxEstimator) {
				maxEstimator = es;
				maxNodeId = graph.vertices[exti].nodeid;
			}
			//printf("");
			free(us);
			free(usT);
			free(usTT);
			free(csEnd);
			free(csFinal);
		}//for  �ڶ��㣬��ÿ���ڵ㵱��Դ�㣬�����ж���������ƽڵ�
		printf("\n��ʵ��Դ��Ϊ%d,���Ƶ�Դ��Ϊ%d\n",realSourceNodeId,maxNodeId);
		if (realSourceNodeId == maxNodeId)
		{
			hit++;
		}
		maxEstimator = -100000;
		maxNodeId = 0;
	}//for��һ�㣬��ÿ��realNode�����ԣ�
	printf("���д���%d\n",hit);
	//getchar();
	printf("�����������......");
	char stop;
	scanf("%s",&stop);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
   hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;
}
hipError_t MatrixWithCuda(float *csFinal,float *csEnd, int height,int width)
{
	const unsigned int mem_size = sizeof(float) * width * height;
    hipError_t cudaStatus;
	float *unitMatrix;//��λ����
	unitMatrix = (float*)malloc(mem_size);
	for(int i=0;i<width;i++)
	{
		for (int j=0;j<height;j++)
		{
			if (i==j)
			{
				unitMatrix[i*width+j]=1.0;
			}
			else
			{
				unitMatrix[i*width+j]=0.0;
			}
		}
	}
	float *d_A;
	float *d_B;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "0cudaSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
	// hipMalloc((void**)&d_idata, mem_size);
    cudaStatus = hipMalloc((void**)&d_A, mem_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "1cudaMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&d_B, mem_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "2cudaMalloc failed!");
        goto Error;
    }
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(d_A, csFinal, mem_size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "3cudaMemcpy failed!");
        goto Error;
    }
	cudaStatus = hipMemcpy(d_B, unitMatrix, mem_size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "3cudaMemcpy failed!");
		goto Error;
	}
    // Launch a kernel on the GPU with one thread for each element.
	//dim3 grid(1, 1);
	dim3 grid((width+BLOCK_DIM-1)/BLOCK_DIM,(width+BLOCK_DIM-1)/BLOCK_DIM);
	dim3 block(BLOCK_DIM, BLOCK_DIM);//����ʱ����16
	for (int k=0;k<width;k++)
	{
		MatrixInverse_Normalized<<<grid,block>>>(d_A,d_B,width,k);
		MatrixInverse_Elimination<<<grid,block>>>(d_A,d_B,width,k);
	}
	
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "4cudaDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(csEnd, d_B, mem_size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "5cudaMemcpy failed!");
        goto Error;
    }
	//float *E = (float *)malloc(mem_size);
	//cudaStatus = hipMemcpy(E, d_A, mem_size, hipMemcpyDeviceToHost);
	////if (cudaStatus != hipSuccess) {
	////	fprintf(stderr, "5cudaMemcpy failed!");
	////	goto Error;
	////}
	////for (int x=0;x<width;x++)
	////{
	////	for (int y=0;y<width;y++)
	////	{
	////		printf("%f ",E[x*width+y]);
	////	}
	////	printf("\n");
	////}
Error:
    hipFree(d_A);
    hipFree(d_B);
	free(unitMatrix);
    return cudaStatus;
}



